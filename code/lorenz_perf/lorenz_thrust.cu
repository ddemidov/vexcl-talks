#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <utility>
#include <cstdlib>

#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include <boost/timer/timer.hpp>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "log.hpp"

namespace odeint = boost::numeric::odeint;

//---------------------------------------------------------------------------
typedef thrust::device_vector<double> state_type;
struct lorenz_system {
    size_t n;
    double sigma, b;
    const state_type &R;

    lorenz_system(size_t n, const state_type &R,
            double sigma = 10.0, double b = 8.0 / 3.0
            ) : n(n), R(R), sigma(sigma), b(b) { }

    struct lorenz_functor {
        double sigma, b;
        lorenz_functor(double sigma, double b)
            : sigma(sigma), b(b) {}

        template<class T>
        __host__ __device__ void operator()(T t) const {
            double x = thrust::get<0>( t );
            double y = thrust::get<1>( t );
            double z = thrust::get<2>( t );
            double R = thrust::get<3>( t );

            thrust::get<4>( t ) = sigma * ( y - x );
            thrust::get<5>( t ) = R * x - y - x * z;
            thrust::get<6>( t ) = -b * z + x * y;
        }
    };

    template<class State, class Deriv>
    void operator()(const State &x, Deriv &dxdt, double t) const {
        auto start = thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin(),
                        x.begin() + n,
                        x.begin() + 2 * n,
                        R.begin(),
                        dxdt.begin(),
                        dxdt.begin() + n,
                        dxdt.begin() + 2 * n
                        ) );

        thrust::for_each(start, start+n, lorenz_functor(sigma, b));
    }
};

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    const size_t n = argc > 1 ? atoi(argv[1]) : 1024;
    const double dt = 0.01;
    const double t_max = 10.0;

    thrust::host_vector<double> r(n);
    const double Rmin = 0.1, Rmax = 50.0;
    for(size_t i = 0; i < n; ++i)
        r[i] = Rmin + i * (Rmax - Rmin) / (n - 1);

    state_type R = r;
    state_type x(3 * n);

    thrust::fill(x.begin(), x.end(), 10.0);

    odeint::runge_kutta4_classic<
        state_type, double, state_type, double,
        odeint::thrust_algebra, odeint::thrust_operations
        > stepper;

    lorenz_system sys(n, R);

#ifndef CPU_RUN
    hipDeviceSynchronize();
#endif
    boost::timer::cpu_timer timer;

    for(double t = 0; t < t_max; t += dt)
        stepper.do_step(sys, x, t, dt);

#ifndef CPU_RUN
    hipDeviceSynchronize();
#endif

    log_perf(argv[0], n, t_max / dt, timer.elapsed());
}
