#include <iostream>
#include <cmath>
#include <utility>
#include <cstdlib>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>

#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <boost/timer.hpp>

#include <hipblas.h>

typedef thrust::device_vector< double > state_type;

hipblasHandle_t cublas_handle;

namespace boost {
namespace numeric {
namespace odeint {

/* only defined for doubles */
struct cublas_operations
{
    //template< class Fac1 , class Fac2 > struct scale_sum2;

    template< class F1 = double , class F2 = F1 >
    struct scale_sum2
    {
        typedef double Fac1;
        typedef double Fac2;
        const Fac1 m_alpha1;
        const Fac2 m_alpha2;

        scale_sum2( const Fac1 alpha1 , const Fac2 alpha2 ) : m_alpha1( alpha1 ) , m_alpha2( alpha2 ) { }

        template< class T1 , class T2 , class T3 >
        void operator()( T1 &t1 , const T2 &t2 , const T3 &t3) const
        {   // t1 = m_alpha1 * t2 + m_alpha2 * t3;
            // we get Containers that have size() and [i]-access

            const int n = t1.size();
            if( thrust::raw_pointer_cast(&(t2[0])) != thrust::raw_pointer_cast(&(t1[0])) )
            {
                hipblasDcopy(cublas_handle, n, thrust::raw_pointer_cast(&(t2[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            }

            hipblasDscal(cublas_handle, n, &m_alpha1, thrust::raw_pointer_cast(&(t1[0])), 1 );
            hipblasDaxpy(cublas_handle, n, &m_alpha2, thrust::raw_pointer_cast(&(t3[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1 );
        }
    };

    template< class F1 = double , class F2 = F1 , class F3 = F2 >
    struct scale_sum3
    {
        typedef double Fac1;
        typedef double Fac2;
        typedef double Fac3;
        const Fac1 m_alpha1;
        const Fac2 m_alpha2;
        const Fac3 m_alpha3;

        scale_sum3( const Fac1 alpha1 , const Fac2 alpha2 , const Fac3 alpha3 )
            : m_alpha1( alpha1 ) , m_alpha2( alpha2 ) , m_alpha3( alpha3 ) { }

        template< class T1 , class T2 , class T3 , class T4 >
        void operator()( T1 &t1 , const T2 &t2 , const T3 &t3 , const T4 &t4 ) const
        {   // t1 = m_alpha1 * t2 + m_alpha2 * t3 + m_alpha3 * t4;
            // we get Containers that have size() and [i]-access

            const int n = t1.size();
            if( thrust::raw_pointer_cast(&(t2[0])) != thrust::raw_pointer_cast(&(t1[0])) )
            {
                hipblasDcopy(cublas_handle, n, thrust::raw_pointer_cast(&(t2[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            }
            hipblasDscal(cublas_handle, n, &m_alpha1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha2, thrust::raw_pointer_cast(&(t3[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha3, thrust::raw_pointer_cast(&(t4[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
        }
    };

    template< class F1 = double , class F2 = F1 , class F3 = F2 , class F4 = F3 >
    struct scale_sum4
    {
        typedef double Fac1;
        typedef double Fac2;
        typedef double Fac3;
        typedef double Fac4;
        const Fac1 m_alpha1;
        const Fac2 m_alpha2;
        const Fac3 m_alpha3;
        const Fac4 m_alpha4;

        scale_sum4( const Fac1 alpha1 , const Fac2 alpha2 , const Fac3 alpha3 , const Fac4 alpha4 )
            : m_alpha1( alpha1 ) , m_alpha2( alpha2 ) , m_alpha3( alpha3 ) , m_alpha4( alpha4 ) { }

        template< class T1 , class T2 , class T3 , class T4 , class T5 >
        void operator()( T1 &t1 , const T2 &t2 , const T3 &t3 , const T4 &t4 , const T5 &t5 ) const
        {   // t1 = m_alpha1 * t2 + m_alpha2 * t3 + m_alpha3 * t4 + m_alpha4 * t5;
            // we get Containers that have size() and [i]-access

            const int n = t1.size();
            if( thrust::raw_pointer_cast(&(t2[0])) != thrust::raw_pointer_cast(&(t1[0])) )
            {
                hipblasDcopy(cublas_handle, n, thrust::raw_pointer_cast(&(t2[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            }

            hipblasDscal(cublas_handle, n, &m_alpha1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha2, thrust::raw_pointer_cast(&(t3[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha3, thrust::raw_pointer_cast(&(t4[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha4, thrust::raw_pointer_cast(&(t5[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
        }
    };


    template< class F1 = double , class F2 = F1 , class F3 = F2 , class F4 = F3 , class F5 = F4 >
    struct scale_sum5
    {
        typedef double Fac1;
        typedef double Fac2;
        typedef double Fac3;
        typedef double Fac4;
        typedef double Fac5;
        const Fac1 m_alpha1;
        const Fac2 m_alpha2;
        const Fac3 m_alpha3;
        const Fac4 m_alpha4;
        const Fac5 m_alpha5;

        scale_sum5( const Fac1 alpha1 , const Fac2 alpha2 , const Fac3 alpha3 , const Fac4 alpha4 , const Fac5 alpha5 )
            : m_alpha1( alpha1 ) , m_alpha2( alpha2 ) , m_alpha3( alpha3 ) , m_alpha4( alpha4 ) , m_alpha5( alpha5 ) { }

        template< class T1 , class T2 , class T3 , class T4 , class T5 , class T6   >
        void operator()( T1 &t1 , const T2 &t2 , const T3 &t3 , const T4 &t4 , const T5 &t5 , const T6 &t6 ) const
        {   // t1 = m_alpha1 * t2 + m_alpha2 * t3 + m_alpha3 * t4 + m_alpha4 * t5 + m_alpha5 * t6;
            // we get Containers that have size() and [i]-access

            const int n = t1.size();
            if( thrust::raw_pointer_cast(&(t2[0])) != thrust::raw_pointer_cast(&(t1[0])) )
            {
                hipblasDcopy(cublas_handle, n, thrust::raw_pointer_cast(&(t2[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            }

            hipblasDscal(cublas_handle, n, &m_alpha1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha2, thrust::raw_pointer_cast(&(t3[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha3, thrust::raw_pointer_cast(&(t4[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha4, thrust::raw_pointer_cast(&(t5[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
            hipblasDaxpy(cublas_handle, n, &m_alpha5, thrust::raw_pointer_cast(&(t6[0])), 1, thrust::raw_pointer_cast(&(t1[0])), 1);
        }
    };

};

} // odeint
} // numeric
} // boost

namespace odeint = boost::numeric::odeint;

const double sigma = 10.0;
const double b     = 8.0 / 3.0;

struct lorenz_system {
    struct lorenz_functor {
        template<class T>
        __host__ __device__ void operator()( T t ) const {
            double x = thrust::get<0>(t);
            double y = thrust::get<1>(t);
            double z = thrust::get<2>(t);
            double r = thrust::get<3>(t);

            thrust::get<4>(t) = sigma * (y - x);
            thrust::get<5>(t) = r * x - y - x * z;
            thrust::get<6>(t) = -b * z + x * y ;
        }
    };

    lorenz_system(size_t N, const state_type &R) : N(N), R(R) { }

    template<class State, class Deriv>
    void operator()(const State &x, Deriv &dxdt, double t) const {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin(x),
                        boost::begin(x) + N,
                        boost::begin(x) + 2 * N,
                        boost::begin(R),
                        boost::begin(dxdt),
                        boost::begin(dxdt) + N,
                        boost::begin(dxdt) + 2 * N ) ),
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin(x) + N,
                        boost::begin(x) + 2 * N,
                        boost::begin(x) + 3 * N,
                        boost::begin(R),
                        boost::begin(dxdt) + N,
                        boost::begin(dxdt) + 2 * N,
                        boost::begin(dxdt) + 3 * N ) ),
                lorenz_functor() );
    }

    size_t N;
    const  state_type &R;
};


const double dt = 0.01;
const double t_max = 100.0;

int main( int argc , char* argv[] ) {
    hipblasCreate(&cublas_handle);

    const size_t n = argc > 1 ? atoi(argv[1]) : 1024;

    std::vector<double> r(n);
    double Rmin = 0.1, Rmax = 50.0, dR = (Rmax - Rmin) / double(n - 1);
    for(size_t i = 0 ; i < n ; ++i)
        r[i] = Rmin + i * dR;

    state_type R = r;
    state_type X(3 * n);

    thrust::fill(X.begin(), X.end(), 10.0);


    odeint::runge_kutta4<
            state_type, double, state_type, double,
            odeint::vector_space_algebra , odeint::cublas_operations
            > stepper;


    lorenz_system sys(n, R);


    hipDeviceSynchronize();
    boost::timer timer;
    odeint::integrate_const(stepper, boost::ref(sys), X, 0.0, t_max, dt);
    hipDeviceSynchronize();
    double time = timer.elapsed();

    std::cout << n << " " << time << std::endl;

    //std::cout << X[0] << std::endl;

    hipblasDestroy(cublas_handle);
}
